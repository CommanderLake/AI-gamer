#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "common.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
__device__ __host__ int div_ceil(int a, int b){ return (a % b != 0) ? (a/b + 1) : (a/b); }
struct pixARGB{
	unsigned char B;
	unsigned char G;
	unsigned char R;
	unsigned char A;
};
struct pixRGB{
	unsigned char B;
	unsigned char G;
	unsigned char R;
};
hiprandGenerator_t gen;
int GS, BS, RPB, CPB, TPG, maxTPB, smemPB;
extern "C" void InitCUDA(){
	const hipError_t cudaRes = hipInit(0);
	if(cudaRes != hipSuccess){
		const char* pStr = nullptr;
		hipDrvGetErrorString(cudaRes, &pStr);
		throw std::runtime_error("CUDA Init failed, error string:\n\n" + std::string(pStr));
	}
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int major;
	hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, 0);
	int minor;
	hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, 0);
	const auto TPM = ConvertSmVer2Cores(major, minor);
	const auto MP = prop.multiProcessorCount;
	const auto warps = prop.warpSize;
	maxTPB = prop.maxThreadsPerBlock;
	smemPB = prop.sharedMemPerBlock;
	GS = warps*MP;
	BS = TPM;
	int TPB = maxTPB;
	TPB = TPB/warps*warps;
	TPG = warps;
	while(TPG*2 <= TPB && TPG < warps){ TPG *= 2; }
	const int groups = TPB/TPG;
	RPB = sqrt(groups);
	CPB = groups/RPB;
	while(RPB*CPB < groups){ if(RPB < CPB){ RPB++; } else{ CPB++; } }
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
}
__global__ void cuARGBtoRGB(const pixARGB* src, pixRGB* dst, int n){
	for(int i = blockIdx.x*blockDim.x + threadIdx.x; i < n; i += blockDim.x*gridDim.x){
		dst[i].R = src[i].R;
		dst[i].G = src[i].G;
		dst[i].B = src[i].B;
	}
}
extern "C" hipError_t ARGBtoRGB(unsigned char* src, unsigned char* dst, int n){
	cuARGBtoRGB<<<GS, BS>>>(reinterpret_cast<pixARGB*>(src), reinterpret_cast<pixRGB*>(dst), n);
	return hipGetLastError();
}
__global__ void cuARGBtoRGBplanar(const unsigned char* src, unsigned char* dst, int n){
	for(int i = blockIdx.x*blockDim.x + threadIdx.x; i < n; i += blockDim.x*gridDim.x){
		const int srcIdx = i*4; // Each pixARGB has 4 bytes
		dst[i] = src[srcIdx + 2]; // R plane
		dst[i + n] = src[srcIdx + 1]; // G plane
		dst[i + 2*n] = src[srcIdx]; // B plane
	}
}
extern "C" hipError_t ARGBtoRGBplanar(unsigned char* src, unsigned char* dst, int n){
	cuARGBtoRGBplanar<<<GS, BS>>>(src, dst, n);
	return hipGetLastError();
}
__device__ float d_loss;
__global__ void mseLossKernel(const __half* predictions, const float* targets, int size){
	extern __shared__ float sdata[];
	const int tid = threadIdx.x;
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	float diff = 0.0f;
	if(idx < size){
		diff = __half2float(predictions[idx]) - targets[idx];
		diff *= diff; // squared error
	}
	sdata[tid] = diff;
	__syncwarp();
	for(int i = 16; i > 0; i >>= 1){
		if(tid < i){ sdata[tid] += sdata[tid + i]; }
		__syncwarp();
	}
	if(tid == 0){ d_loss = sdata[0]; }
}
extern "C" float MseLoss(const __half* d_predictions, const float* d_targets, int size){
	auto gridSize = div_ceil(size, BS);
	mseLossKernel<<<gridSize, BS, BS*sizeof(float)>>>(d_predictions, d_targets, size);
	float h_loss;
	hipMemcpyFromSymbol(&h_loss, HIP_SYMBOL(d_loss), sizeof(float));
	return h_loss/size;
}
__global__ void convertAndNormalizeKernel(__half* output, unsigned char* input, size_t size){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < size){ output[idx] = __float2half(static_cast<float>(input[idx])/255.0f); }
}
extern "C" void ConvertAndNormalize(__half* output, unsigned char* input, size_t size){
	auto gridSize = div_ceil(size, BS);
	convertAndNormalizeKernel<<<gridSize, BS>>>(output, input, size);
}
__global__ void UnConvertAndUnNormalizeKernel(unsigned char* output, const __half* input, size_t size){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < size){ output[idx] = static_cast<unsigned char>(__half2float(input[idx])*255.0f); }
}
extern "C" void UnConvertAndUnNormalize(unsigned char* output, const __half* input, size_t size){
	auto gridSize = div_ceil(size, BS);
	UnConvertAndUnNormalizeKernel<<<gridSize, BS>>>(output, input, size);
}
__global__ void convertFloatToHalfKernel(float* src, __half* dst, size_t n){
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < n){ dst[i] = __float2half(src[i]); }
}
extern "C" void ConvertFloatToHalf(float* src, __half* dst, size_t n){
	auto gridSize = div_ceil(n, BS);
	convertFloatToHalfKernel<<<gridSize, BS>>>(src, dst, n);
}
__global__ void convertHalfToFloatKernel(__half* src, float* dst, size_t n){
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < n){ dst[i] = __half2float(src[i]); }
}
extern "C" void ConvertHalfToFloat(__half* src, float* dst, size_t n){
	auto gridSize = div_ceil(n, BS);
	convertHalfToFloatKernel<<<gridSize, BS>>>(src, dst, n);
}
__global__ void HeInitKernel(__half* halfWeights, float* weights, int n, float scale){
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < n){ halfWeights[i] = __float2half(weights[i]*scale); }
}
extern "C" void HeInit(__half* weightHalf, int numWeights, float fanIn){
	float* weightFloat;
	hipMalloc(&weightFloat, numWeights*sizeof(float));
	hiprandGenerateNormal(gen, weightFloat, numWeights, 0.0f, 1.0f);
	auto gridSize = div_ceil(numWeights, BS);
	HeInitKernel<<<gridSize, BS>>>(weightHalf, weightFloat, numWeights, sqrtf(2.0f/fanIn));
	hipFree(weightFloat);
}
__global__ void sgdHalfKernel(__half* param, const float learningRate, const __half* gradParam, const int n){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < n){ param[idx] -= __float2half(learningRate*__half2float(gradParam[idx])); }
}
extern "C" void SGDHalf(__half* param, const float learningRate, const __half* gradParam, const int size){
	auto gridSize = div_ceil(size, BS);
	sgdHalfKernel<<<gridSize, BS>>>(param, learningRate, gradParam, size);
}
__global__ void sgdFloatKernel(float* param, const float learningRate, const float* gradParam, const int n){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < n){ param[idx] -= learningRate*gradParam[idx]; }
}
extern "C" void SGDFloat(float* param, const float learningRate, const float* gradParam, const int size){
	auto gridSize = div_ceil(size, BS);
	sgdFloatKernel<<<gridSize, BS>>>(param, learningRate, gradParam, size);
}
__device__ const float beta1F = 0.9f;
__device__ const float beta2F = 0.999f;
__device__ const float epsilonF = 1e-8f;
__global__ void adamKernelHalf(__half* param, float* m, float* v, const float learningRate, const __half* gradParam, const int n, const int t){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < n){
		const float grad = __half2float(gradParam[idx]);
		float param_value = __half2float(param[idx]);
		m[idx] = beta1F*m[idx] + (1.0f - beta1F)*grad;
		v[idx] = beta2F*v[idx] + (1.0f - beta2F)*grad*grad;
		const float m_hat = m[idx]/(1.0f - powf(beta1F, t));
		const float v_hat = v[idx]/(1.0f - powf(beta2F, t));
		param_value -= learningRate*m_hat/(sqrtf(v_hat) + epsilonF);
		param[idx] = __float2half(param_value);
	}
}
void AdamHalf(__half* param, float* m, float* v, const float learningRate, const __half* gradParam, const int size, const int t){
	auto gridSize = div_ceil(size, BS);
	adamKernelHalf<<<gridSize, BS>>>(param, m, v, learningRate, gradParam, size, t);
}
__global__ void adamKernelFloat(float* param, float* m, float* v, const float learningRate, const float* gradParam, const int n, const int t){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < n){
		const float grad = gradParam[idx];
		float param_value = param[idx];
		m[idx] = beta1F*m[idx] + (1.0f - beta1F)*grad;
		v[idx] = beta2F*v[idx] + (1.0f - beta2F)*grad*grad;
		const float m_hat = m[idx]/(1.0f - powf(beta1F, t));
		const float v_hat = v[idx]/(1.0f - powf(beta2F, t));
		param_value -= learningRate*m_hat/(sqrtf(v_hat) + epsilonF);
		param[idx] = param_value;
	}
}
void AdamFloat(float* param, float* m, float* v, const float learningRate, const float* gradParam, const int size, const int t){
	auto gridSize = div_ceil(size, BS);
	adamKernelFloat<<<gridSize, BS>>>(param, m, v, learningRate, gradParam, size, t);
}
__global__ void adamWKernelHalf(__half* param, float* m, float* v, const float learningRate, const __half* gradParam, const int n, const int t, const float weightDecay){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < n){
		const float grad = __half2float(gradParam[idx]);
		float param_value = __half2float(param[idx]);
		m[idx] = beta1F*m[idx] + (1.0f - beta1F)*grad;
		v[idx] = beta2F*v[idx] + (1.0f - beta2F)*grad*grad;
		const float m_hat = m[idx]/(1.0f - powf(beta1F, t));
		const float v_hat = v[idx]/(1.0f - powf(beta2F, t));
		param_value -= learningRate*(m_hat / (sqrtf(v_hat) + epsilonF));
		param_value -= learningRate*weightDecay*param_value;
		param[idx] = __float2half(param_value);
	}
}
void AdamWHalf(__half* param, float* m, float* v, const float learningRate, const __half* gradParam, const int size, const int t, const float weightDecay){
	auto gridSize = div_ceil(size, BS);
	adamWKernelHalf<<<gridSize, BS>>>(param, m, v, learningRate, gradParam, size, t, weightDecay);
}
__global__ void adamWKernelFloat(float* param, float* m, float* v, const float learningRate, const float* gradParam, const int n, const int t, const float weightDecay){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < n){
		const float grad = gradParam[idx];
		float param_value = param[idx];
		m[idx] = beta1F*m[idx] + (1.0f - beta1F)*grad;
		v[idx] = beta2F*v[idx] + (1.0f - beta2F)*grad*grad;
		const float m_hat = m[idx]/(1.0f - powf(beta1F, t));
		const float v_hat = v[idx]/(1.0f - powf(beta2F, t));
		param_value -= learningRate*(m_hat / (sqrtf(v_hat) + epsilonF));
		param_value -= learningRate*weightDecay*param_value;
		param[idx] = param_value;
	}
}
void AdamWFloat(float* param, float* m, float* v, const float learningRate, const float* gradParam, const int size, const int t, const float weightDecay){
	auto gridSize = div_ceil(size, BS);
	adamWKernelFloat<<<gridSize, BS>>>(param, m, v, learningRate, gradParam, size, t, weightDecay);
}
__global__ void warp8SmemKernel(int N, int K, const half*__restrict__ A, const half*__restrict__ B, half*__restrict__ C, int BS, int TPG, int CPB){
	extern __shared__ half A_smem[];
	const int A_smem_iters = div_ceil(K, BS);
#pragma unroll
	for(int i = 0; i < A_smem_iters; ++i){
		const int idx = i*BS + threadIdx.x;
		A_smem[idx] = A[idx];
	}
	__syncthreads();
	const int group_id = threadIdx.x/TPG;
	const int group_col = blockIdx.x*CPB + group_id;
	if(group_col >= N){ return; }
	const int K_iters = div_ceil(K, TPG);
	const int group_lane_id = threadIdx.x % TPG;
	float tmp = 0.0;
#pragma unroll
	for(int i = 0; i < K_iters; ++i){
		const int A_idx = i*TPG + group_lane_id;
		const int B_idx = i*TPG + group_lane_id + group_col*K;
		tmp += __half2float(A_smem[A_idx])*__half2float(B[B_idx]);
	}
	constexpr unsigned int mask = 0xffffffff;
#pragma unroll
	for(int i = TPG/2; i >= 1; i /= 2){ tmp += __shfl_xor_sync(mask, tmp, i); }
	if(group_lane_id == 0){ C[group_col] = __float2half(tmp); }
}
extern "C" void Hgemv(const half* A, const half* B, half* C, int N, int K){
	static int smem_max_size = K*sizeof(half);
	dim3 block(BS);
	dim3 grid(div_ceil(N, CPB));
	warp8SmemKernel<<<grid, block, smem_max_size>>>(N, K, A, B, C, BS, TPG, CPB);
}
__global__ void gemmHFFKernel(bool transA, bool transB, int M, int N, int K, const half*__restrict__ A, int lda, const float*__restrict__ B, int ldb, float*__restrict__ C, int ldc, int BS, int TPG, int RPB, int CPB){
	extern __shared__ half A_smem[];
	const int A_smem_iters = div_ceil(transA ? M : K, BS);
#pragma unroll
	for(int i = 0; i < A_smem_iters; ++i){
		const int idx = i*BS + threadIdx.x;
		if(idx < (transA ? M : K)){ A_smem[idx] = A[transA ? (idx*lda) : (idx)]; }
	}
	__syncthreads();
	const int group_id = threadIdx.x/TPG;
	const int group_row = blockIdx.y*RPB + group_id;
	const int group_col = blockIdx.x*CPB + group_id;
	if(group_row >= M || group_col >= N){ return; }
	const int K_iters = div_ceil(K, TPG);
	const int group_lane_id = threadIdx.x % TPG;
	float tmp = 0.0f;
#pragma unroll
	for(int i = 0; i < K_iters; ++i){
		const int k = i*TPG + group_lane_id;
		if(k < K){
			const int A_idx = transA ? (k*M + group_row) : (group_row*K + k);
			const int B_idx = transB ? (group_col*ldb + k) : (k*ldb + group_col);
			tmp += __half2float(A_smem[A_idx])*B[B_idx];
		}
	}
	constexpr unsigned int mask = 0xffffffff;
#pragma unroll
	for(int i = TPG/2; i >= 1; i /= 2){ tmp += __shfl_xor_sync(mask, tmp, i); }
	if(group_lane_id == 0){ C[group_row*ldc + group_col] = tmp; }
}
extern "C" void GemmHff(int M, int N, int K, bool transA, bool transB, const half* A, int lda, const float* B, int ldb, float* C, int ldc){
	int smem_size = (transA ? M : K)*sizeof(half);
	dim3 block(BS);
	dim3 grid(div_ceil(N, CPB), div_ceil(M, RPB));
	gemmHFFKernel<<<grid, block, smem_size>>>(transA, transB, M, N, K, A, lda, B, ldb, C, ldc, BS, TPG, RPB, CPB);
}
__global__ void gradientKernel(__half* gradients, const __half* predictions, const float* targets, float delta, int size, float scale){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < size){
		const float diff = __half2float(predictions[idx]) - targets[idx];
		float gradient;
		if(fabs(diff) <= delta){
			gradient = diff;
		} else{
			gradient = delta*((diff > 0) - (diff < 0));
		}
		gradients[idx] = __float2half(gradient*scale);
	}
}
extern "C" void Gradient(__half* d_gradient, const __half* d_predictions, const float* d_targets, int batchSize, int n, float scale){
	gradientKernel<<<n, batchSize>>>(d_gradient, d_predictions, d_targets, 1.0f, n*batchSize, scale);
}
__global__ void biasGradientsKernel(const __half* gradInput, __half* gradBias, int c, int batchSize){
	extern __shared__ float sharedGrad[];
	const int channelIdx = blockIdx.x*blockDim.x + threadIdx.x;
	const int idxInBlock = threadIdx.x;
	if(channelIdx < c){
		float sum = 0.0f;
		for(int i = 0; i < batchSize; i++){ sum += __half2float(gradInput[i*c + channelIdx]); }
		sharedGrad[idxInBlock] = sum;
	} else{ sharedGrad[idxInBlock] = 0.0f; }
	__syncthreads();
	// Reduce sum in shared memory
	for(int stride = blockDim.x/2; stride > 0; stride >>= 1){
		if(idxInBlock < stride){ sharedGrad[idxInBlock] += sharedGrad[idxInBlock + stride]; }
		__syncthreads();
	}
	if(idxInBlock == 0){
		// Write the reduced sum to the global bias gradient
		for(int i = 0; i < blockDim.x && blockIdx.x*blockDim.x + i < c; i++){ gradBias[blockIdx.x*blockDim.x + i] = __float2half(sharedGrad[i]); }
	}
}
extern "C" void BiasGradient(const __half* gradInput, __half* gradBias, int c, int batchSize){
	auto gridSize = div_ceil(c, BS);
	size_t sharedMemSize = BS*sizeof(float);
	biasGradientsKernel<<<gridSize, BS, sharedMemSize>>>(gradInput, gradBias, c, batchSize);
}
__global__ void clipGradsKernel(__half* grad, int size, float lower, float upper){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < size){
		const float gradValue = __half2float(grad[idx]);
		if(isnan(gradValue)){
			grad[idx] = __float2half(1e-6f);
		} else if(gradValue >= 0.0f && gradValue < lower){
			grad[idx] = __float2half(lower);
		} else if(gradValue <= -0.0f && gradValue > -lower){
			grad[idx] = __float2half(-lower);
		} else if(gradValue > upper){
			grad[idx] = __float2half(upper);
		} else if(gradValue < -upper){
			grad[idx] = __float2half(-upper);
		}
	}
}
extern "C" void ClipGrads(__half* grad, int size){
	auto gridSize = div_ceil(size, BS);
	clipGradsKernel<<<gridSize, BS>>>(grad, size, 1e-6f, 128.0f);
}
__global__ void scaleKernel(__half* data, int size, float scale){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < size){ data[idx] = data[idx]*__float2half(scale); }
}
extern "C" void Scale(__half* data, int size, float scale){
	auto gridSize = div_ceil(size, BS);
	scaleKernel<<<gridSize, BS>>>(data, size, scale);
}
__global__ void leakyReluKernel(__half* data, int size, __half negativeSlope){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < size){ if(data[idx] < __half(0)){ data[idx] *= negativeSlope; } }
}
extern "C" void LeakyRelu(__half* data, int size, float negativeSlope){
	auto gridSize = div_ceil(size, BS);
	leakyReluKernel<<<gridSize, BS>>>(data, size, __float2half(negativeSlope));
}
__global__ void leakyReluBackwardKernel(__half* gradient, const __half* inData, int size, __half negativeSlope){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < size){ if(inData[idx] < __half(0.0f)){ gradient[idx] *= negativeSlope; } }
}
extern "C" void LeakyReluBackward(__half* gradient, const __half* inData, int size, float negativeSlope){
	auto gridSize = div_ceil(size, BS);
	leakyReluBackwardKernel<<<gridSize, BS>>>(gradient, inData, size, __float2half(negativeSlope));
}
__global__ void sigmoidForwardKernel(__half* data, int numSigmoidOutputs, int batchSize, int outputSize){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < batchSize*outputSize){
		if(idx % outputSize < numSigmoidOutputs){
			const float val = __half2float(data[idx]);
			const float sigmoid = 1.0f/(1.0f + expf(-val));
			data[idx] = __float2half(sigmoid);
		}
	}
}
extern "C" void SigmoidForward(__half* data, int numSigmoidOutputs, int batchSize, int outputSize){
	int numBlocks = div_ceil(batchSize*outputSize, BS);
	sigmoidForwardKernel<<<numBlocks, BS>>>(data, numSigmoidOutputs, batchSize, outputSize);
}
__global__ void sigmoidBackwardKernel(__half* grad, const __half* data, int numSigmoidOutputs, int batchSize, int outputSize){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < batchSize*outputSize){
		if(idx % outputSize < numSigmoidOutputs){
			const float val = __half2float(data[idx]);
			grad[idx] = __float2half(__half2float(grad[idx])*val*(1.0f - val));
		}
	}
}
extern "C" void SigmoidBackward(__half* grad, const __half* data, int numSigmoidOutputs, int batchSize, int outputSize){
	int numBlocks = div_ceil(batchSize*outputSize, BS);
	sigmoidBackwardKernel<<<numBlocks, BS>>>(grad, data, numSigmoidOutputs, batchSize, outputSize);
}
__global__ void computeMeanVarianceKernel(const __half* data, float* mean, float* variance, int N, int C, int HW){
	extern __shared__ float sdata[];
	float* s_sum = sdata;
	float* s_sq_sum = &sdata[blockDim.x];
	int tid = threadIdx.x;
	int cid = blockIdx.x; // Changed from blockIdx.y to blockIdx.x
	if(cid >= C) return; // Guard against excessive blocks
	float thread_sum = 0.0f;
	float thread_sq_sum = 0.0f;
	for(int n = 0; n < N; ++n){
		for(int i = tid; i < HW; i += blockDim.x){
			int idx = (n*C + cid)*HW + i;
			if(idx < N*C*HW){
				// Boundary check
				float val = __half2float(data[idx]);
				thread_sum += val;
				thread_sq_sum += val*val;
			}
		}
	}
	s_sum[tid] = thread_sum;
	s_sq_sum[tid] = thread_sq_sum;
	__syncthreads();
	// Parallel reduction in shared memory
	for(unsigned int s = blockDim.x/2; s > 0; s >>= 1){
		if(tid < s){
			s_sum[tid] += s_sum[tid + s];
			s_sq_sum[tid] += s_sq_sum[tid + s];
		}
		__syncthreads();
	}
	if(tid == 0){
		int total_elements = N*HW;
		mean[cid] = s_sum[0]/total_elements;
		variance[cid] = fmaxf(s_sq_sum[0]/total_elements - mean[cid]*mean[cid], 0.0f);
	}
}
__global__ void layerNormForwardKernel(__half* output, const __half* data, const float* gamma, const float* beta, const float* mean, const float* variance, int N, int C, int HW, float epsilon){
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	int cid = blockIdx.y;
	if(cid >= C) return; // Guard against excessive blocks
	if(tid < N*HW){
		int idx = (cid*N + tid/HW)*HW + tid % HW;
		if(idx < N*C*HW){
			// Boundary check
			float x = __half2float(data[idx]);
			float norm = (x - mean[cid])/sqrtf(variance[cid] + epsilon);
			output[idx] = __float2half(norm*gamma[cid] + beta[cid]);
		}
	}
}
extern "C" void LayerNormForward(__half* output, const __half* data, const float* gamma, const float* beta, float* mean, float* variance, int N, int C, int HW, float epsilon){
	dim3 gridDim((C + TPG - 1)/TPG, 1, 1); // Ensure we cover all channels
	dim3 blockDim(TPG, 1, 1);
	int sharedMemSize = 2*TPG*sizeof(float);
	computeMeanVarianceKernel<<<gridDim, blockDim, sharedMemSize>>>(data, mean, variance, N, C, HW);
	// Check for kernel launch errors
	hipError_t err = hipGetLastError();
	if(err != hipSuccess){
		printf("computeMeanVarianceKernel launch failed: %s\n", hipGetErrorString(err));
		return;
	}
	gridDim = dim3((N*HW + TPG - 1)/TPG, C, 1);
	layerNormForwardKernel<<<gridDim, blockDim>>>(output, data, gamma, beta, mean, variance, N, C, HW, epsilon);
	// Check for kernel launch errors
	err = hipGetLastError();
	if(err != hipSuccess){
		printf("layerNormForwardKernel launch failed: %s\n", hipGetErrorString(err));
		return;
	}
	// Synchronize to catch any errors during kernel execution
	hipDeviceSynchronize();
	err = hipGetLastError();
	if(err != hipSuccess){ printf("Kernel execution failed: %s\n", hipGetErrorString(err)); }
}
__global__ void layerNormBackwardKernel(__half* gradIn, const __half* gradOut, const __half* data, const float* gamma, float* gradGamma, float* gradBeta, const float* mean, const float* variance, int N, int C, int HW, const float epsilon){
	extern __shared__ float sdata[];
	float* sGradGamma = sdata;
	float* sGradBeta = &sdata[blockDim.x];
	const int tid = threadIdx.x;
	const int cid = blockIdx.y;
	float threadGradGamma = 0.0f;
	float threadGradBeta = 0.0f;
	const float invStd = rsqrtf(variance[cid] + epsilon);
	for(int n = 0; n < N; ++n){
		for(int i = tid; i < HW; i += blockDim.x){
			const int idx = (cid*N + n)*HW + i;
			const float x = __half2float(data[idx]);
			const float dy = __half2float(gradOut[idx]);
			const float xHat = (x - mean[cid])*invStd;
			threadGradGamma += xHat*dy;
			threadGradBeta += dy;
			if(gradIn != nullptr){
				const float dx = gamma[cid]*invStd*(dy - (xHat*threadGradGamma + threadGradBeta)/(N*HW));
				gradIn[idx] = __float2half(dx);
			}
		}
	}
	sGradGamma[tid] = threadGradGamma;
	sGradBeta[tid] = threadGradBeta;
	__syncthreads();
	// Parallel reduction in shared memory
	for(unsigned int s = blockDim.x/2; s > 0; s >>= 1){
		if(tid < s){
			sGradGamma[tid] += sGradGamma[tid + s];
			sGradBeta[tid] += sGradBeta[tid + s];
		}
		__syncthreads();
	}
	if(tid == 0){
		atomicAdd(&gradGamma[cid], sGradGamma[0]);
		atomicAdd(&gradBeta[cid], sGradBeta[0]);
	}
}
extern "C" void LayerNormBackward(__half* gradIn, const __half* gradOut, const __half* data, const float* gamma, float* gradGamma, float* gradBeta, const float* mean, const float* variance, int N, int C, int HW, const float epsilon){
	dim3 gridDim(1, C, 1);
	dim3 blockDim(TPG, 1, 1);
	int sharedMemSize = 2*TPG*sizeof(float);
	layerNormBackwardKernel<<<gridDim, blockDim, sharedMemSize>>>(gradIn, gradOut, data, gamma, gradGamma, gradBeta, mean, variance, N, C, HW, epsilon);
}
__device__ bool deviceResult;
__global__ void isNaNKernel(__half* data, int size){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ bool foundNaN;
	if(threadIdx.x == 0){ foundNaN = false; }
	__syncthreads();
	if(idx < size && __hisnan(data[idx])){ foundNaN = true; }
	__syncthreads();
	if(threadIdx.x == 0 && foundNaN){ deviceResult = true; }
}
extern "C" bool isnanHalf(__half* data, int size){
	bool h_result = false;
	hipMemcpyToSymbol(HIP_SYMBOL(deviceResult), &h_result, sizeof(bool));
	auto gridSize = div_ceil(size, BS);
	isNaNKernel<<<gridSize, BS>>>(data, size);
	hipMemcpyFromSymbol(&h_result, HIP_SYMBOL(deviceResult), sizeof(bool));
	return h_result;
}
#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "common.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
//#define COLS_PER_BLOCK 32    // COLS_PER_WARP*WARPS_PER_BLOCK
//#define THREADS_PER_GROUP 4  // WARP_SIZE/COLS_PER_WARP
__device__ __host__ int div_ceil(int a, int b){
	return (a % b != 0) ? (a/b + 1) : (a/b);
}
struct pixARGB{
	unsigned char B;
	unsigned char G;
	unsigned char R;
	unsigned char A;
};
struct pixRGB{
	unsigned char B;
	unsigned char G;
	unsigned char R;
};
__global__ void cuARGBtoRGB(pixARGB* src, pixRGB* dst, int n){
	for(int i = blockIdx.x*blockDim.x + threadIdx.x; i < n; i += blockDim.x*gridDim.x){
		dst[i].R = src[i].R;
		dst[i].G = src[i].G;
		dst[i].B = src[i].B;
	}
}
__global__ void cuARGBtoRGBplanar(unsigned char* src, unsigned char* dst, int n){
	for(int i = blockIdx.x*blockDim.x + threadIdx.x; i < n; i += blockDim.x*gridDim.x){
		const int srcIdx = i*4; // Each pixARGB has 4 bytes
		dst[i] = src[srcIdx + 2]; // R plane
		dst[i + n] = src[srcIdx + 1]; // G plane
		dst[i + 2*n] = src[srcIdx]; // B plane
	}
}
hiprandGenerator_t gen;
int GS, BS, RPB, CPB, TPG, maxTPB, smemPB;
extern "C" void InitCUDA(){
	const hipError_t cudaRes = hipInit(0);
	if(cudaRes != hipSuccess){
		const char* pStr = nullptr;
		hipDrvGetErrorString(cudaRes, &pStr);
		throw std::runtime_error("CUDA Init failed, error string:\n\n" + std::string(pStr));
	}
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);  // Assuming we're using device 0
	int major;
	hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, 0);
	int minor;
	hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, 0);
	const auto TPM = _ConvertSMVer2Cores(major, minor);
	const auto MP = prop.multiProcessorCount;
	const auto warps = prop.warpSize;
	maxTPB = prop.maxThreadsPerBlock;
	smemPB = prop.sharedMemPerBlock;
	GS = warps*MP;
	BS = TPM;
	int TPB = maxTPB;
	TPB = TPB/warps*warps;
	TPG = warps;
	while(TPG*2 <= TPB && TPG < warps){
		TPG *= 2;
	}
	const int groups = TPB/TPG;
	RPB = sqrt(groups);
	CPB = groups/RPB;
	while(RPB*CPB < groups){
		if(RPB < CPB){
			RPB++;
		} else{
			CPB++;
		}
	}
	hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
}
extern "C" hipError_t ARGBtoRGB(hipDeviceptr_t src, hipDeviceptr_t dst, int n){
	cuARGBtoRGB<<<GS, BS>>>(reinterpret_cast<pixARGB*>(src), reinterpret_cast<pixRGB*>(dst), n);
	return hipGetLastError();
}
extern "C" hipError_t ARGBtoRGBplanar(hipDeviceptr_t src, hipDeviceptr_t dst, int n){
	cuARGBtoRGBplanar<<<GS, BS>>>(reinterpret_cast<unsigned char*>(src), reinterpret_cast<unsigned char*>(dst), n);
	return hipGetLastError();
}

__device__ float d_loss;
__global__ void mseLossKernel(const __half* predictions, const float* targets, int size){
	extern __shared__ float sdata[];
	const int tid = threadIdx.x;
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	float diff = 0.0f;
	if(idx < size){
		diff = __half2float(predictions[idx]) - targets[idx];
		diff *= diff; // squared error
	}
	sdata[tid] = diff;
	__syncwarp();
	for(int i = 16; i > 0; i >>= 1){
		if(tid < i){
			sdata[tid] += sdata[tid + i];
		}
		__syncwarp();
	}
	if(tid == 0){
		d_loss = sdata[0];
	}
}
extern "C" float mseLoss(const __half* d_predictions, const float* d_targets, int size){
	auto gridSize = div_ceil(size, BS);
	mseLossKernel<<<gridSize, BS, BS*sizeof(float)>>>(d_predictions, d_targets, size);
	float h_loss;
	hipMemcpyFromSymbol(&h_loss, HIP_SYMBOL(d_loss), sizeof(float));
	return h_loss / size;
}

__global__ void convertAndNormalizeKernel(unsigned char* input, __half* output, size_t size){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < size){
		output[idx] = __float2half(static_cast<float>(input[idx])/255.0f);
	}
}
extern "C" void convertAndNormalize(unsigned char* input, __half* output, size_t size){
	auto gridSize = div_ceil(size, BS);
	convertAndNormalizeKernel<<<gridSize, BS>>>(input, output, size);
}

__global__ void convertFloatToHalfKernel(float* src, __half* dst, size_t n){
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < n){
		dst[i] = __float2half(src[i]);
	}
}
extern "C" void convertFloatToHalf(float* src, __half* dst, size_t n){
	auto gridSize = div_ceil(n, BS);
	convertFloatToHalfKernel<<<gridSize, BS>>>(src, dst, n);
}

__global__ void convertHalfToFloatKernel(__half* src, float* dst, size_t n){
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < n){
		dst[i] = __half2float(src[i]);
	}
}
extern "C" void convertHalfToFloat(__half* src, float* dst, size_t n){
	auto gridSize = div_ceil(n, BS);
	convertHalfToFloatKernel<<<gridSize, BS>>>(src, dst, n);
}

__global__ void HeInitKernel(__half* halfWeights, float* weights, int n, float scale){
	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i < n){
		halfWeights[i] = __float2half(weights[i]*scale);
	}
}
extern "C" void HeInit(__half* weightHalf, int numWeights, float fanIn){
	float* weightFloat;
	hipMalloc(&weightFloat, numWeights*sizeof(float));
	hiprandGenerateNormal(gen, weightFloat, numWeights, 0.0f, 1.0f);
	auto gridSize = div_ceil(numWeights, BS);
	HeInitKernel<<<gridSize, BS>>>(weightHalf, weightFloat, numWeights, sqrtf(2.0f / fanIn));
	hipFree(weightFloat);
}

__global__ void sgdHalfKernel(__half* param, const float learningRate, const float* gradParam, const int n){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < n){
		param[idx] += __float2half(learningRate*gradParam[idx]);
	}
}
extern "C" void SGDHalf(__half* param, const float learningRate, const float* gradParam, const int size){
	auto gridSize = div_ceil(size, BS);
	sgdHalfKernel<<<gridSize, BS>>>(param, learningRate, gradParam, size);
}

__global__ void sgdFloatKernel(float* param, const float learningRate, const float* gradParam, const int n){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < n){
		param[idx] += learningRate*gradParam[idx];
	}
}
extern "C" void SGDFloat(float* param, const float learningRate, const float* gradParam, const int size){
	auto gridSize = div_ceil(size, BS);
	sgdFloatKernel<<<gridSize, BS>>>(param, learningRate, gradParam, size);
}

__device__ const float beta1F = 0.9f;
__device__ const float beta2F = 0.999f;
__device__ const float epsilonF = 1e-6f;
__global__ void adamKernelHalf(__half* param, float* m, float* v, const float learningRate, const __half* gradParam, const int n, const int t){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < n){
		const float grad = __half2float(gradParam[idx]);
		float param_value = __half2float(param[idx]);
		m[idx] = beta1F*m[idx] + (1.0f - beta1F)*grad;
		v[idx] = beta2F*v[idx] + (1.0f - beta2F)*grad*grad;
		const float m_hat = m[idx]/(1.0f - powf(beta1F, t));
		const float v_hat = v[idx]/(1.0f - powf(beta2F, t));
		param_value -= learningRate*m_hat/(sqrtf(v_hat) + epsilonF);
		param[idx] = __float2half(param_value);
	}
}
void AdamHalf(__half* param, float* m, float* v, const float learningRate, const __half* gradParam, const int size, const int t){
	auto gridSize = div_ceil(size, BS);
	adamKernelHalf<<<gridSize, BS>>>(param, m, v, learningRate, gradParam, size, t);
}

__global__ void adamKernelFloat(float* param, float* m, float* v, const float learningRate, const float* gradParam, const int n, const int t){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < n){
		const float grad = gradParam[idx];
		float param_value = param[idx];
		m[idx] = beta1F*m[idx] + (1.0f - beta1F)*grad;
		v[idx] = beta2F*v[idx] + (1.0f - beta2F)*grad*grad;
		const float m_hat = m[idx]/(1.0f - powf(beta1F, t));
		const float v_hat = v[idx]/(1.0f - powf(beta2F, t));
		param_value -= learningRate*m_hat/(sqrtf(v_hat) + epsilonF);
		param[idx] = param_value;
	}
}
void AdamFloat(float* param, float* m, float* v, const float learningRate, const float* gradParam, const int size, const int t){
	auto gridSize = div_ceil(size, BS);
	adamKernelFloat<<<gridSize, BS>>>(param, m, v, learningRate, gradParam, size, t);
}

__global__ void adamWKernelHalf(__half* param, float* m, float* v, const float learningRate, const __half* gradParam, const int n, const int t, const float weightDecay){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < n){
		const float grad = __half2float(gradParam[idx]);
		float param_value = __half2float(param[idx]);
		m[idx] = beta1F*m[idx] + (1.0f - beta1F)*grad;
		v[idx] = beta2F*v[idx] + (1.0f - beta2F)*grad*grad;
		const float m_hat = m[idx]/(1.0f - powf(beta1F, t));
		const float v_hat = v[idx]/(1.0f - powf(beta2F, t));
		param_value -= learningRate*(m_hat / (sqrtf(v_hat) + epsilonF) + weightDecay*param_value);
		param[idx] = __float2half(param_value);
	}
}
void AdamWHalf(__half* param, float* m, float* v, const float learningRate, const __half* gradParam, const int size, const int t, const float weightDecay){
	auto gridSize = div_ceil(size, BS);
	adamWKernelHalf<<<gridSize, BS>>>(param, m, v, learningRate, gradParam, size, t, weightDecay);
}

__global__ void adamWKernelFloat(float* param, float* m, float* v, const float learningRate, const float* gradParam, const int n, const int t, const float weightDecay){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < n){
		const float grad = gradParam[idx];
		float param_value = param[idx];
		m[idx] = beta1F*m[idx] + (1.0f - beta1F)*grad;
		v[idx] = beta2F*v[idx] + (1.0f - beta2F)*grad*grad;
		const float m_hat = m[idx]/(1.0f - powf(beta1F, t));
		const float v_hat = v[idx]/(1.0f - powf(beta2F, t));
		param_value -= learningRate*(m_hat / (sqrtf(v_hat) + epsilonF) + weightDecay*param_value);
		param[idx] = param_value;
	}
}
void AdamWFloat(float* param, float* m, float* v, const float learningRate, const float* gradParam, const int size, const int t, const float weightDecay){
	auto gridSize = div_ceil(size, BS);
	adamWKernelFloat<<<gridSize, BS>>>(param, m, v, learningRate, gradParam, size, t, weightDecay);
}

__global__ void warp8SmemKernel(int N, int K, const half *__restrict__ A, const half *__restrict__ B, half *__restrict__ C, int BS, int TPG, int CPB){
	extern __shared__ half A_smem[];
	const int A_smem_iters = div_ceil(K, BS);
#pragma unroll
	for(int i = 0; i < A_smem_iters; ++i){
		const int idx = i*BS + threadIdx.x;
		A_smem[idx] = A[idx];
	}
	__syncthreads();
	const int group_id = threadIdx.x/TPG;
	const int group_col = blockIdx.x*CPB + group_id;
	if(group_col >= N){
		return;
	}
	const int K_iters = div_ceil(K, TPG);
	const int group_lane_id = threadIdx.x % TPG;
	float tmp = 0.0;
#pragma unroll
	for(int i = 0; i < K_iters; ++i){
		const int A_idx = i*TPG + group_lane_id;
		const int B_idx = i*TPG + group_lane_id + group_col*K;
		tmp += __half2float(A_smem[A_idx])*__half2float(B[B_idx]);
	}
	constexpr unsigned int mask = 0xffffffff;
#pragma unroll
	for(int i = TPG/2; i >= 1; i /= 2){
		tmp += __shfl_xor_sync(mask, tmp, i);
	}
	if(group_lane_id == 0){
		C[group_col] = __float2half(tmp);
	}
}
extern "C" void Hgemv(const half *A, const half *B, half *C, int N, int K){
	static int smem_max_size = K*sizeof(half);
	dim3 block(BS);
	dim3 grid(div_ceil(N, CPB));
	warp8SmemKernel<<<grid, block, smem_max_size>>>(N, K, A, B, C, BS, TPG, CPB);
}

__global__ void gemmHFFKernel(bool transA, bool transB, int M, int N, int K, const half *__restrict__ A, int lda, const float *__restrict__ B, int ldb, float *__restrict__ C, int ldc, int BS, int TPG, int RPB, int CPB){
	extern __shared__ half A_smem[];
	const int A_smem_iters = div_ceil(transA ? M : K, BS);
#pragma unroll
	for(int i = 0; i < A_smem_iters; ++i){
		const int idx = i*BS + threadIdx.x;
		if(idx < (transA ? M : K)){
			A_smem[idx] = A[transA ? (idx * lda) : (idx)];
		}
	}
	__syncthreads();
	const int group_id = threadIdx.x / TPG;
	const int group_row = blockIdx.y*RPB + group_id;
	const int group_col = blockIdx.x*CPB + group_id;
	if(group_row >= M || group_col >= N){
		return;
	}
	const int K_iters = div_ceil(K, TPG);
	const int group_lane_id = threadIdx.x % TPG;
	float tmp = 0.0f;
#pragma unroll
	for(int i = 0; i < K_iters; ++i){
		const int k = i*TPG + group_lane_id;
		if(k < K){
			const int A_idx = transA ? (k*M + group_row) : (group_row*K + k);
			const int B_idx = transB ? (group_col*ldb + k) : (k*ldb + group_col);
			tmp += __half2float(A_smem[A_idx])*B[B_idx];
		}
	}
	constexpr unsigned int mask = 0xffffffff;
#pragma unroll
	for(int i = TPG/2; i >= 1; i /= 2){
		tmp += __shfl_xor_sync(mask, tmp, i);
	}
	if(group_lane_id == 0){
		C[group_row*ldc + group_col] = tmp;
	}
}
extern "C" void gemmHFF(int M, int N, int K, bool transA, bool transB, const half *A, int lda, const float *B, int ldb, float *C, int ldc){
	int smem_size = (transA ? M : K)*sizeof(half);
	dim3 block(BS);
	dim3 grid(div_ceil(N, CPB), div_ceil(M, RPB));
	gemmHFFKernel<<<grid, block, smem_size>>>(transA, transB, M, N, K, A, lda, B, ldb, C, ldc, BS, TPG, RPB, CPB);
}

__global__ void gradientKernel(__half* gradients, const __half* predictions, const float* targets, float delta, int size){
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < size){
		const float diff = __half2float(predictions[idx]) - targets[idx];
		if(fabs(diff) <= delta){
			gradients[idx] = __float2half(diff);
		} else{
			gradients[idx] = __float2half(delta*((diff > 0) - (diff < 0)));
		}
	}
}
extern "C" void gradient(__half* d_gradient, const __half* d_predictions, const float* d_targets, int batchSize, int n){
	gradientKernel<<<n, batchSize>>>(d_gradient, d_predictions, d_targets, 1.0f, n*batchSize);
}

__global__ void biasGradientsKernel(const __half* gradInput, __half* gradBias, int c, int batchSize){
	extern __shared__ float sharedGrad[];
	const int channelIdx = blockIdx.x*blockDim.x + threadIdx.x;
	const int idxInBlock = threadIdx.x;
	if(channelIdx < c){
		float sum = 0.0f;
		for(int i = 0; i < batchSize; i++){ sum += __half2float(gradInput[i*c + channelIdx]); }
		sharedGrad[idxInBlock] = sum;
	} else{ sharedGrad[idxInBlock] = 0.0f; }
	__syncthreads();
	// Reduce sum in shared memory
	for(int stride = blockDim.x / 2; stride > 0; stride >>= 1){
		if(idxInBlock < stride){ sharedGrad[idxInBlock] += sharedGrad[idxInBlock + stride]; }
		__syncthreads();
	}
	if(idxInBlock == 0){
		// Write the reduced sum to the global bias gradient
		for(int i = 0; i < blockDim.x && blockIdx.x*blockDim.x + i < c; i++){ gradBias[blockIdx.x*blockDim.x + i] = __float2half(sharedGrad[i]); }
	}
}
extern "C" void biasGradient(const __half* gradInput, __half* gradBias, int c, int batchSize){
	auto gridSize = div_ceil(c, BS);
	size_t sharedMemSize = BS*sizeof(float);
	biasGradientsKernel<<<gridSize, BS, sharedMemSize>>>(gradInput, gradBias, c, batchSize);
}

__global__ void clipGradsKernel(__half* grad, int size, float lower, float upper){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < size){
		const float gradValue = __half2float(grad[idx]);
		if(gradValue < lower && gradValue >= 0.0f){
			grad[idx] = __float2half(lower);
		} else if(gradValue > -lower && gradValue <= -0.0f){
			grad[idx] = __float2half(-lower);
		} else if(gradValue > upper){
			grad[idx] = __float2half(upper);
		} else if(gradValue < -upper){
			grad[idx] = __float2half(-upper);
		} else if(isnan(gradValue)){
			grad[idx] = __float2half(1e-6f);
		}
	}
}
extern "C" void clipGrads(__half* grad, int size){
	auto gridSize = div_ceil(size, BS);
	clipGradsKernel<<<gridSize, BS>>>(grad, size, 1e-6f, 1.0f);
}

__global__ void scaleKernel(__half* data, int size, float scale){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < size){
		data[idx] = data[idx]*__float2half(scale);
	}
}
extern "C" void scale(__half* data, int size, float scale){
	auto gridSize = div_ceil(size, BS);
	scaleKernel<<<gridSize, BS>>>(data, size, scale);
}

__global__ void leakyReluKernel(__half* data, int size, __half negativeSlope){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < size){
		if(data[idx] < __half(0)){
			data[idx] *= negativeSlope;
		}
	}
}
extern "C" void leakyRelu(__half* data, int size, float negativeSlope){
	auto gridSize = div_ceil(size, BS);
	leakyReluKernel<<<gridSize, BS>>>(data, size, __float2half(negativeSlope));
}

__global__ void leakyReluBackwardKernel(const __half* gradIn, const __half* inData, __half* gradOut, int size, __half negativeSlope){
	const int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(idx < size){
		if(inData[idx] > __half(0)){
			gradOut[idx] = gradIn[idx];
		} else{
			gradOut[idx] = gradIn[idx]*negativeSlope;
		}
	}
}
extern "C" void leakyReluBackward(const __half* gradIn, const __half* inData, __half* gradOut, int size, float negativeSlope){
	auto gridSize = div_ceil(size, BS);
	leakyReluBackwardKernel<<<gridSize, BS>>>(gradIn, inData, gradOut, size, __float2half(negativeSlope));
}